#include "hip/hip_runtime.h"
#include "miner/cuda/miner.h"

#include "miner/common/constants.h"

#include <boost/assert.hpp>
#include <boost/log/trivial.hpp>
#include <cgbn/cgbn.h>
#include <gmpxx.h>

// NEEDS TO BE INCLUDED AFTER CGBN AND GMP
#include "cu_helpers.h"

#include <cassert>
#include <iostream>
#include <vector>

using namespace miner::common;
using namespace miner::cuda;

namespace kernel
{

static const uint32_t BIT_SIZE = 256;
static const uint32_t C_SIZE = 219;

typedef cgbn_mem_t<BIT_SIZE> bn_mem_t;

__constant__ bn_mem_t global_mimc_p;
__constant__ bn_mem_t global_mimc_c[C_SIZE];

struct MimcParams
{
    bn_mem_t P;
    bn_mem_t *C;
    std::size_t C_size;
};

struct CudaWorkItem
{
    int64_t x;
    int64_t y;
    bool is_planet;
    bn_mem_t hash;
};

template <uint32_t tpi> struct BnParams
{
    static const uint32_t TPI = tpi; // GCBN threads per intstance.
};

} // namespace kernel

struct miner::cuda::CachedDeviceMemory
{
    uint32_t side_length;
    std::size_t bytes_size;
    kernel::CudaWorkItem *d_batch;
    kernel::CudaWorkItem *h_batch;
};

namespace kernel
{

void to_mpz(mpz_t r, const bn_mem_t &x)
{
    mpz_import(r, BIT_SIZE / 32, -1, sizeof(uint32_t), 0, 0, x._limbs);
}

void from_mpz(mpz_srcptr s, bn_mem_t &n)
{
    uint32_t count = BIT_SIZE / 32;
    uint32_t *x = n._limbs;
    size_t words;

    if (mpz_sizeinbase(s, 2) > count * 32)
    {
        fprintf(stderr, "from_mpz failed -- result does not fit\n");
        exit(1);
    }

    mpz_export(x, &words, -1, sizeof(uint32_t), 0, 0, s);
    while (words < count)
    {
        x[words++] = 0;
    }
}

template <class env_t, class bn_t = typename env_t::cgbn_t>
__device__ __forceinline__ void wrap_coordinate(env_t &env, bn_t &c_bn, int64_t c, const bn_t &p)
{
    typename env_t::cgbn_t n, m;
    if (c >= 0)
    {
        uint32_t c_ = static_cast<uint32_t>(c);
        env.set_ui32(n, c_);
        env.set(c_bn, n);
        return;
    }
    uint32_t c_ = static_cast<uint32_t>(-c);
    env.set_ui32(m, c_);
    env.sub(n, p, m);
    env.set(c_bn, n);
}

template <class env_t, class bn_t = typename env_t::cgbn_t>
__device__ __forceinline__ void field_add(env_t &env, bn_t &r, const bn_t &a, const bn_t &b, const bn_t &p)
{
    env.add(r, a, b);
    while (env.compare(r, p) > 0)
    {
        env.sub(r, r, p);
    }
}

template <class env_t, class bn_t = typename env_t::cgbn_t, class bn_wide_t = typename env_t::cgbn_wide_t>
__device__ __forceinline__ void field_mul(env_t &env, bn_t &r, const bn_t &a, const bn_t &b, const bn_t &p)
{
    bn_wide_t w;
    env.mul_wide(w, a, b);
    env.rem_wide(r, w, p);
}

template <class env_t, class bn_t = typename env_t::cgbn_t>
__device__ __forceinline__ void fifth_power(env_t &env, bn_t &r, const bn_t &n, bn_t &s, bn_t &f, const bn_t &p)
{
    field_mul(env, s, n, n, p);
    field_mul(env, f, s, s, p);
    field_mul(env, r, f, n, p);
}

template <class env_t, class bn_t = typename env_t::cgbn_t> class Sponge
{
  public:
    __device__ void reset(env_t &env)
    {
        env.set_ui32(l_, 0);
        env.set_ui32(r_, 0);
    }

    __device__ void inject(env_t &env, const bn_t &x, const bn_t &P)
    {
        field_add(env, l_, l_, x, P);
    }

    __device__ void mix(env_t &env, const bn_t &key, const bn_t *C, std::size_t C_size, const bn_t &P)
    {
        for (uint32_t j = 0; j < C_size; ++j)
        {
            field_add(env, t0_, key, l_, P);
            field_add(env, t1_, t0_, C[j], P);
            fifth_power(env, t0_, t1_, t2_, t3_, P);
            field_add(env, t1_, t0_, r_, P);
            env.set(r_, l_);
            env.set(l_, t1_);
        }
        field_add(env, t0_, key, l_, P);
        fifth_power(env, t1_, t0_, t2_, t3_, P);
        field_add(env, t0_, t1_, r_, P);
        env.set(r_, t0_);
    }

    __device__ void save(env_t &env)
    {
        env.set(snap_l_, l_);
        env.set(snap_r_, r_);
    }

    __device__ void restore(env_t &env)
    {
        env.set(l_, snap_l_);
        env.set(r_, snap_r_);
    }

    __device__ void result(env_t &env, bn_t &out)
    {
        env.set(out, l_);
    }

  private:
    bn_t l_;
    bn_t r_;

    bn_t snap_l_;
    bn_t snap_r_;

    bn_t t0_;
    bn_t t1_;
    bn_t t2_;
    bn_t t3_;
    bn_t t4_;
};

template <class bn_params>
__global__ void mine_batch_kernel(cgbn_error_report_t *report, const ChunkFootprint chunk, CudaWorkItem *result,
                                  uint32_t items_per_thread, bn_mem_t planet_threshold_mem, bn_mem_t key_mem)
{
    using context_t = cgbn_context_t<bn_params::TPI>;
    using env_t = cgbn_env_t<context_t, BIT_SIZE>;
    using bn_t = typename env_t::cgbn_t;

    // coord x is fixed for each thread
    // we know that blockDim = 1
    if (blockIdx.x >= chunk.side_length)
    {
        return;
    }

    int64_t coord_x = chunk.bottom_left.x + blockIdx.x;

    context_t ctx(cgbn_report_monitor, report);
    env_t env(ctx);

    // Copy mimc constants to memory
    bn_t C[C_SIZE];
    for (uint32_t i = 0; i < C_SIZE; ++i)
    {
        env.load(C[i], &(global_mimc_c[i]));
    }

    Sponge<env_t> sponge;
    bn_t P, key, planet_threshold;
    bn_t yi, xi, hash;
    env.load(P, &global_mimc_p);
    env.load(key, &key_mem);
    env.load(planet_threshold, &planet_threshold_mem);

    __syncthreads();

    wrap_coordinate(env, xi, coord_x, P);
    sponge.reset(env);
    sponge.inject(env, xi, P);
    sponge.mix(env, key, C, C_SIZE, P);
    sponge.save(env);

    uint32_t start_size_y =
        blockIdx.y * (blockDim.y + items_per_thread) + items_per_thread * (threadIdx.x / bn_params::TPI);
    for (uint32_t i = 0; i < items_per_thread; ++i)
    {
        if (start_size_y + i >= chunk.side_length)
        {
            break;
        }

        int64_t coord_y = chunk.bottom_left.y + start_size_y + i;

        sponge.restore(env);
        wrap_coordinate(env, yi, coord_y, P);
        sponge.inject(env, yi, P);
        sponge.mix(env, key, C, C_SIZE, P);
        sponge.result(env, hash);

        uint32_t result_idx = blockIdx.x + chunk.side_length * (start_size_y + i);
        result[result_idx].is_planet = env.compare(hash, planet_threshold) < 0;
        if (result[result_idx].is_planet)
        {
            env.store(&(result[result_idx].hash), hash);
        }
        result[result_idx].x = coord_x;
        result[result_idx].y = coord_y;
    }
}

template <class bn_params>
void run_mine_batch(int device, const CudaMinerOptions &options, const ChunkFootprint &chunk,
                    const std::shared_ptr<CachedDeviceMemory> &cache_, const bn_mem_t &planet_threshold,
                    const bn_mem_t &key, std::vector<PlanetLocation> &result)
{
    cgbn_error_report_t *bn_report;

    CUDA_CHECK(hipSetDevice(device));
    CUDA_CHECK(cgbn_error_report_alloc(&bn_report));

    uint32_t items_per_block = options.thread_work_size * options.block_size;
    // grid_size_y = ceil(side_length / items_per_block)
    uint32_t grid_size_y = (chunk.side_length + items_per_block - 1) / items_per_block;
    dim3 block_size(bn_params::TPI * options.block_size, 1);
    dim3 grid_size(chunk.side_length, grid_size_y);

    BOOST_LOG_TRIVIAL(info) << "Starting miner kernel";
    BOOST_LOG_TRIVIAL(info) << "  CUDA configuration:";
    BOOST_LOG_TRIVIAL(info) << "  -      bottom left: "
                            << "(" << chunk.bottom_left.x << ", " << chunk.bottom_left.y << ")";
    BOOST_LOG_TRIVIAL(info) << "  -      side length: " << chunk.side_length;
    BOOST_LOG_TRIVIAL(info) << "  - thread_work_size: " << options.thread_work_size;
    BOOST_LOG_TRIVIAL(info) << "  -       block_size: " << options.block_size;
    BOOST_LOG_TRIVIAL(info) << "  -           BN TPI: " << bn_params::TPI;
    BOOST_LOG_TRIVIAL(info) << "  - final block size: "
                            << "(" << block_size.x << ", " << block_size.y << ")";
    BOOST_LOG_TRIVIAL(info) << "  -  final grid size: "
                            << "(" << grid_size.x << ", " << grid_size.y << ")";

    mine_batch_kernel<bn_params>
        <<<grid_size, block_size>>>(bn_report, chunk, cache_->d_batch, options.thread_work_size, planet_threshold, key);

    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipGetLastError());
    CGBN_CHECK(bn_report);

    CUDA_CHECK(hipMemcpy(cache_->h_batch, cache_->d_batch, cache_->bytes_size, hipMemcpyDeviceToHost));

    mpz_class planet_hash;
    CudaWorkItem *cpu_batch = cache_->h_batch;
    for (std::size_t i = 0; i < chunk.side_length * chunk.side_length; ++i)
    {
        if (cpu_batch[i].is_planet)
        {
            to_mpz(planet_hash.get_mpz_t(), cpu_batch[i].hash);
            Coordinate coord(cpu_batch[i].x, cpu_batch[i].y);
            std::string hash = planet_hash.get_str();
            PlanetLocation location(std::move(coord), std::move(hash));
            result.push_back(location);
        }
    }
}

} // namespace kernel

CudaMiner::CudaMiner(int device, const CudaMinerOptions &options)
    : device_(device), options_(options), initialized_(false), cache_(nullptr)
{
}

CudaMiner::~CudaMiner()
{
}

void CudaMiner::initialize()
{
    if (initialized_)
    {
        return;
    }

    initialized_ = true;

    // load P and C on device
    kernel::bn_mem_t P_bn;
    kernel::bn_mem_t C_bn[kernel::C_SIZE];

    BOOST_ASSERT(C.size() == kernel::C_SIZE);
    kernel::from_mpz(P.get_mpz_t(), P_bn);
    for (std::size_t i = 0; i < C.size(); i++)
    {
        kernel::from_mpz(C[i].get_mpz_t(), C_bn[i]);
    }

    CUDA_CHECK(hipSetDevice(device_));

    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(kernel::global_mimc_p), &P_bn, sizeof(kernel::bn_mem_t)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(kernel::global_mimc_c), C_bn, sizeof(kernel::bn_mem_t) * C.size()));
}

void CudaMiner::prepare_cache(uint32_t side_length)
{
    if (cache_ != nullptr && cache_->side_length == side_length)
    {
        BOOST_LOG_TRIVIAL(debug) << "CUDA cached data already present";
        return;
    }

    // free up old memory, if any
    if (cache_ != nullptr)
    {
        free(cache_->h_batch);
        CUDA_CHECK(hipFree(cache_->d_batch));
    }
    else
    {
        cache_ = std::make_shared<CachedDeviceMemory>();
    }

    std::size_t batch_size = side_length * side_length;
    std::size_t bytes_size = sizeof(kernel::CudaWorkItem) * batch_size;

    cache_->h_batch = static_cast<kernel::CudaWorkItem *>(malloc(bytes_size));

    CUDA_CHECK(hipSetDevice(device_));
    CUDA_CHECK(hipMalloc(&cache_->d_batch, bytes_size));
    cache_->side_length = side_length;
    cache_->bytes_size = bytes_size;
}

void CudaMiner::mine(const ChunkFootprint &chunk, uint32_t rarity, uint32_t key, std::vector<PlanetLocation> &result)
{
    kernel::bn_mem_t planet_threshold_bn, key_bn;

    initialize();
    prepare_cache(chunk.side_length);

    mpz_class rarity_mpz(rarity);
    mpz_class planet_threshold = P / rarity_mpz;
    kernel::from_mpz(planet_threshold.get_mpz_t(), planet_threshold_bn);

    mpz_class key_mpz(key);
    kernel::from_mpz(key_mpz.get_mpz_t(), key_bn);

    switch (options_.threads_per_item)
    {
    case ThreadsPerItem::TPI_4:
        return kernel::run_mine_batch<kernel::BnParams<4>>(device_, options_, chunk, cache_, planet_threshold_bn,
                                                           key_bn, result);
    case ThreadsPerItem::TPI_8:
        return kernel::run_mine_batch<kernel::BnParams<8>>(device_, options_, chunk, cache_, planet_threshold_bn,
                                                           key_bn, result);
    case ThreadsPerItem::TPI_16:
        return kernel::run_mine_batch<kernel::BnParams<16>>(device_, options_, chunk, cache_, planet_threshold_bn,
                                                            key_bn, result);
    default:
        return kernel::run_mine_batch<kernel::BnParams<32>>(device_, options_, chunk, cache_, planet_threshold_bn,
                                                            key_bn, result);
    }
}